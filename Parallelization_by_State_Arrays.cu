#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <fstream>
#include <windows.h>

using namespace std;

uint8_t sBoxO[256] =
{
    0x63, 0x7C, 0x77, 0x7B, 0xF2, 0x6B, 0x6F, 0xC5, 0x30, 0x01, 0x67, 0x2B, 0xFE, 0xD7, 0xAB, 0x76,
    0xCA, 0x82, 0xC9, 0x7D, 0xFA, 0x59, 0x47, 0xF0, 0xAD, 0xD4, 0xA2, 0xAF, 0x9C, 0xA4, 0x72, 0xC0,
    0xB7, 0xFD, 0x93, 0x26, 0x36, 0x3F, 0xF7, 0xCC, 0x34, 0xA5, 0xE5, 0xF1, 0x71, 0xD8, 0x31, 0x15,
    0x04, 0xC7, 0x23, 0xC3, 0x18, 0x96, 0x05, 0x9A, 0x07, 0x12, 0x80, 0xE2, 0xEB, 0x27, 0xB2, 0x75,
    0x09, 0x83, 0x2C, 0x1A, 0x1B, 0x6E, 0x5A, 0xA0, 0x52, 0x3B, 0xD6, 0xB3, 0x29, 0xE3, 0x2F, 0x84,
    0x53, 0xD1, 0x00, 0xED, 0x20, 0xFC, 0xB1, 0x5B, 0x6A, 0xCB, 0xBE, 0x39, 0x4A, 0x4C, 0x58, 0xCF,
    0xD0, 0xEF, 0xAA, 0xFB, 0x43, 0x4D, 0x33, 0x85, 0x45, 0xF9, 0x02, 0x7F, 0x50, 0x3C, 0x9F, 0xA8,
    0x51, 0xA3, 0x40, 0x8F, 0x92, 0x9D, 0x38, 0xF5, 0xBC, 0xB6, 0xDA, 0x21, 0x10, 0xFF, 0xF3, 0xD2,
    0xCD, 0x0C, 0x13, 0xEC, 0x5F, 0x97, 0x44, 0x17, 0xC4, 0xA7, 0x7E, 0x3D, 0x64, 0x5D, 0x19, 0x73,
    0x60, 0x81, 0x4F, 0xDC, 0x22, 0x2A, 0x90, 0x88, 0x46, 0xEE, 0xB8, 0x14, 0xDE, 0x5E, 0x0B, 0xDB,
    0xE0, 0x32, 0x3A, 0x0A, 0x49, 0x06, 0x24, 0x5C, 0xC2, 0xD3, 0xAC, 0x62, 0x91, 0x95, 0xE4, 0x79,
    0xE7, 0xC8, 0x37, 0x6D, 0x8D, 0xD5, 0x4E, 0xA9, 0x6C, 0x56, 0xF4, 0xEA, 0x65, 0x7A, 0xAE, 0x08,
    0xBA, 0x78, 0x25, 0x2E, 0x1C, 0xA6, 0xB4, 0xC6, 0xE8, 0xDD, 0x74, 0x1F, 0x4B, 0xBD, 0x8B, 0x8A,
    0x70, 0x3E, 0xB5, 0x66, 0x48, 0x03, 0xF6, 0x0E, 0x61, 0x35, 0x57, 0xB9, 0x86, 0xC1, 0x1D, 0x9E,
    0xE1, 0xF8, 0x98, 0x11, 0x69, 0xD9, 0x8E, 0x94, 0x9B, 0x1E, 0x87, 0xE9, 0xCE, 0x55, 0x28, 0xDF,
    0x8C, 0xA1, 0x89, 0x0D, 0xBF, 0xE6, 0x42, 0x68, 0x41, 0x99, 0x2D, 0x0F, 0xB0, 0x54, 0xBB, 0x16
};

uint8_t rcon[256] = {
    0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a,
    0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39,
    0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a,
    0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8,
    0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef,
    0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc,
    0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b,
    0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3,
    0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94,
    0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20,
    0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35,
    0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f,
    0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04,
    0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63,
    0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd,
    0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d
};

void rotWord(uint8_t* word);
void subWord(uint8_t* word);
void rCon(uint8_t* word, int iteration);
void keyExpansion(uint8_t* inputKey, uint8_t* expandedKeys);

double PCFreq = 0.0;
__int64 CounterStart = 0;

void StartCounter()
{
    LARGE_INTEGER li;
    if (!QueryPerformanceFrequency(&li))
        cout << "QueryPerformanceFrequency failed!\n";

    PCFreq = double(li.QuadPart) / 1000.0;

    QueryPerformanceCounter(&li);
    CounterStart = li.QuadPart;
}

double GetCounter()
{
    LARGE_INTEGER li;
    QueryPerformanceCounter(&li);
    return double(li.QuadPart - CounterStart) / PCFreq;
}

void rotWord(uint8_t* word) {
    uint8_t temp = word[0];
    for (int i = 0; i < 3; i++) {
        word[i] = word[i + 1];
    }
    word[3] = temp;
}

void subWord(uint8_t* word) {
    for (int i = 0; i < 4; i++) {
        word[i] = sBoxO[word[i]];
    }
}

void rCon(uint8_t* word, int iteration) {
    word[0] ^= rcon[iteration];
}

void keyExpansion(uint8_t* inputKey, uint8_t* expandedKeys) {
    for (int i = 0; i < 32; i++) {
        expandedKeys[i] = inputKey[i];
    }

    int bytesGenerated = 32;
    int rconIteration = 1;
    uint8_t tmpCore[4];

    while (bytesGenerated < 240) {

        for (int i = 0; i < 4; i++) {
            tmpCore[i] = expandedKeys[i + bytesGenerated - 4];
        }

        if (bytesGenerated % 32 == 0) {
            rotWord(tmpCore);
            subWord(tmpCore);
            rCon(tmpCore, rconIteration);
            rconIteration++;
        }
        else if (bytesGenerated % 16 == 0) {
            subWord(tmpCore);
        }

        for (int i = 0; i < 4; i++) {
            expandedKeys[bytesGenerated] = expandedKeys[bytesGenerated - 16] ^ tmpCore[i];
            bytesGenerated++;
        }

    }
}

__device__ void load(uint8_t* sBox, uint8_t* mixMatrix) {

    sBox[0] = 0x63; sBox[1] = 0x7c; sBox[2] = 0x77; sBox[3] = 0x7b; sBox[4] = 0xf2; sBox[5] = 0x6b; sBox[6] = 0x6f; sBox[7] = 0xc5; sBox[8] = 0x30; sBox[9] = 0x1; sBox[10] = 0x67; sBox[11] = 0x2b; sBox[12] = 0xfe; sBox[13] = 0xd7; sBox[14] = 0xab; sBox[15] = 0x76;
    sBox[16] = 0xca; sBox[17] = 0x82; sBox[18] = 0xc9; sBox[19] = 0x7d; sBox[20] = 0xfa; sBox[21] = 0x59; sBox[22] = 0x47; sBox[23] = 0xf0; sBox[24] = 0xad; sBox[25] = 0xd4; sBox[26] = 0xa2; sBox[27] = 0xaf; sBox[28] = 0x9c; sBox[29] = 0xa4; sBox[30] = 0x72; sBox[31] = 0xc0;
    sBox[32] = 0xb7; sBox[33] = 0xfd; sBox[34] = 0x93; sBox[35] = 0x26; sBox[36] = 0x36; sBox[37] = 0x3f; sBox[38] = 0xf7; sBox[39] = 0xcc; sBox[40] = 0x34; sBox[41] = 0xa5; sBox[42] = 0xe5; sBox[43] = 0xf1; sBox[44] = 0x71; sBox[45] = 0xd8; sBox[46] = 0x31; sBox[47] = 0x15;
    sBox[48] = 0x4; sBox[49] = 0xc7; sBox[50] = 0x23; sBox[51] = 0xc3; sBox[52] = 0x18; sBox[53] = 0x96; sBox[54] = 0x5; sBox[55] = 0x9a; sBox[56] = 0x7; sBox[57] = 0x12; sBox[58] = 0x80; sBox[59] = 0xe2; sBox[60] = 0xeb; sBox[61] = 0x27; sBox[62] = 0xb2; sBox[63] = 0x75;
    sBox[64] = 0x9; sBox[65] = 0x83; sBox[66] = 0x2c; sBox[67] = 0x1a; sBox[68] = 0x1b; sBox[69] = 0x6e; sBox[70] = 0x5a; sBox[71] = 0xa0; sBox[72] = 0x52; sBox[73] = 0x3b; sBox[74] = 0xd6; sBox[75] = 0xb3; sBox[76] = 0x29; sBox[77] = 0xe3; sBox[78] = 0x2f; sBox[79] = 0x84;
    sBox[80] = 0x53; sBox[81] = 0xd1; sBox[82] = 0x0; sBox[83] = 0xed; sBox[84] = 0x20; sBox[85] = 0xfc; sBox[86] = 0xb1; sBox[87] = 0x5b; sBox[88] = 0x6a; sBox[89] = 0xcb; sBox[90] = 0xbe; sBox[91] = 0x39; sBox[92] = 0x4a; sBox[93] = 0x4c; sBox[94] = 0x58; sBox[95] = 0xcf;
    sBox[96] = 0xd0; sBox[97] = 0xef; sBox[98] = 0xaa; sBox[99] = 0xfb; sBox[100] = 0x43; sBox[101] = 0x4d; sBox[102] = 0x33; sBox[103] = 0x85; sBox[104] = 0x45; sBox[105] = 0xf9; sBox[106] = 0x2; sBox[107] = 0x7f; sBox[108] = 0x50; sBox[109] = 0x3c; sBox[110] = 0x9f; sBox[111] = 0xa8;
    sBox[112] = 0x51; sBox[113] = 0xa3; sBox[114] = 0x40; sBox[115] = 0x8f; sBox[116] = 0x92; sBox[117] = 0x9d; sBox[118] = 0x38; sBox[119] = 0xf5; sBox[120] = 0xbc; sBox[121] = 0xb6; sBox[122] = 0xda; sBox[123] = 0x21; sBox[124] = 0x10; sBox[125] = 0xff; sBox[126] = 0xf3; sBox[127] = 0xd2;
    sBox[128] = 0xcd; sBox[129] = 0xc; sBox[130] = 0x13; sBox[131] = 0xec; sBox[132] = 0x5f; sBox[133] = 0x97; sBox[134] = 0x44; sBox[135] = 0x17; sBox[136] = 0xc4; sBox[137] = 0xa7; sBox[138] = 0x7e; sBox[139] = 0x3d; sBox[140] = 0x64; sBox[141] = 0x5d; sBox[142] = 0x19; sBox[143] = 0x73;
    sBox[144] = 0x60; sBox[145] = 0x81; sBox[146] = 0x4f; sBox[147] = 0xdc; sBox[148] = 0x22; sBox[149] = 0x2a; sBox[150] = 0x90; sBox[151] = 0x88; sBox[152] = 0x46; sBox[153] = 0xee; sBox[154] = 0xb8; sBox[155] = 0x14; sBox[156] = 0xde; sBox[157] = 0x5e; sBox[158] = 0xb; sBox[159] = 0xdb;
    sBox[160] = 0xe0; sBox[161] = 0x32; sBox[162] = 0x3a; sBox[163] = 0xa; sBox[164] = 0x49; sBox[165] = 0x6; sBox[166] = 0x24; sBox[167] = 0x5c; sBox[168] = 0xc2; sBox[169] = 0xd3; sBox[170] = 0xac; sBox[171] = 0x62; sBox[172] = 0x91; sBox[173] = 0x95; sBox[174] = 0xe4; sBox[175] = 0x79;
    sBox[176] = 0xe7; sBox[177] = 0xc8; sBox[178] = 0x37; sBox[179] = 0x6d; sBox[180] = 0x8d; sBox[181] = 0xd5; sBox[182] = 0x4e; sBox[183] = 0xa9; sBox[184] = 0x6c; sBox[185] = 0x56; sBox[186] = 0xf4; sBox[187] = 0xea; sBox[188] = 0x65; sBox[189] = 0x7a; sBox[190] = 0xae; sBox[191] = 0x8;
    sBox[192] = 0xba; sBox[193] = 0x78; sBox[194] = 0x25; sBox[195] = 0x2e; sBox[196] = 0x1c; sBox[197] = 0xa6; sBox[198] = 0xb4; sBox[199] = 0xc6; sBox[200] = 0xe8; sBox[201] = 0xdd; sBox[202] = 0x74; sBox[203] = 0x1f; sBox[204] = 0x4b; sBox[205] = 0xbd; sBox[206] = 0x8b; sBox[207] = 0x8a;
    sBox[208] = 0x70; sBox[209] = 0x3e; sBox[210] = 0xb5; sBox[211] = 0x66; sBox[212] = 0x48; sBox[213] = 0x3; sBox[214] = 0xf6; sBox[215] = 0xe; sBox[216] = 0x61; sBox[217] = 0x35; sBox[218] = 0x57; sBox[219] = 0xb9; sBox[220] = 0x86; sBox[221] = 0xc1; sBox[222] = 0x1d; sBox[223] = 0x9e;
    sBox[224] = 0xe1; sBox[225] = 0xf8; sBox[226] = 0x98; sBox[227] = 0x11; sBox[228] = 0x69; sBox[229] = 0xd9; sBox[230] = 0x8e; sBox[231] = 0x94; sBox[232] = 0x9b; sBox[233] = 0x1e; sBox[234] = 0x87; sBox[235] = 0xe9; sBox[236] = 0xce; sBox[237] = 0x55; sBox[238] = 0x28; sBox[239] = 0xdf;
    sBox[240] = 0x8c; sBox[241] = 0xa1; sBox[242] = 0x89; sBox[243] = 0xd; sBox[244] = 0xbf; sBox[245] = 0xe6; sBox[246] = 0x42; sBox[247] = 0x68; sBox[248] = 0x41; sBox[249] = 0x99; sBox[250] = 0x2d; sBox[251] = 0xf; sBox[252] = 0xb0; sBox[253] = 0x54; sBox[254] = 0xbb; sBox[255] = 0x16;


    mixMatrix[0] = 2; mixMatrix[1] = 3; mixMatrix[2] = 1; mixMatrix[3] = 1; mixMatrix[4] = 1; mixMatrix[5] = 2; mixMatrix[6] = 3; mixMatrix[7] = 1; mixMatrix[8] = 1;
    mixMatrix[9] = 1; mixMatrix[10] = 2; mixMatrix[11] = 3; mixMatrix[12] = 3; mixMatrix[13] = 1; mixMatrix[14] = 1; mixMatrix[15] = 2;

}

__device__ void mixColumns(uint8_t* state, uint8_t* mixMatrix) {

    uint8_t tempResult = 0;
    uint8_t pval = 0;
    uint8_t currState;
    uint8_t result[16];

    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            for (int k = 0; k < 4; k++) {
                currState = state[k * 4 + j];
                switch (mixMatrix[i * 4 + k]) {
                case 1:
                    tempResult = currState;
                    break;
                case 2:
                    if (currState >= 128) {
                        tempResult = currState << 1;
                        tempResult ^= 27;
                    }
                    else {
                        tempResult = currState << 1;
                    }
                    break;
                case 3:
                    if (currState >= 128) {
                        tempResult = currState << 1;
                        tempResult ^= 27;
                    }
                    else {
                        tempResult = currState << 1;
                    }
                    tempResult ^= currState;
                    break;
                }
                pval ^= tempResult;
            }
            result[i * 4 + j] = pval;
        }

    }

    for (int i = 0; i < 16; i++) {
        state[i] = result[i];
    }
}

__device__ void addRoundKey(uint8_t* state, uint8_t* key, int round) {

    for (int i = 0; i < 16; i++) {
        state[i] ^= key[round * 16 + i];
    }

}

__device__ void subBytes(uint8_t* state, uint8_t* sBox) {

    for (int i = 0; i < 16; i++) {
        state[i] = sBox[state[i]];
    }

}

__device__ void shiftRows(uint8_t* state) {

    uint8_t out[16];

    out[4] = state[5];
    out[5] = state[6];
    out[6] = state[7];
    out[7] = state[4];

    out[8] = state[10];
    out[9] = state[11];
    out[10] = state[8];
    out[11] = state[9];

    out[12] = state[15];
    out[13] = state[12];
    out[14] = state[13];
    out[15] = state[14];

    for (int i = 4; i < 16; i++) {
        state[i] = out[i];
    }
}

__global__ void AESEncryption(uint8_t* text, uint8_t* key, int blockNumber) {

    int thread = blockDim.x * blockIdx.x + threadIdx.x;

    __shared__ uint8_t sBox[256];
    __shared__ uint8_t mixMatrix[16];

    if (thread < blockNumber) {

        if (thread == 0) {
            load(sBox, mixMatrix);
        }
        __syncthreads();

        uint8_t currBlock[16];
        for (int i = 0; i < 16; i++) {
            currBlock[i] = text[thread * 16 + i];
        }
        addRoundKey(currBlock, key, 0);

        for (int i = 1; i < 14; i++) {
            subBytes(currBlock, sBox);
            shiftRows(currBlock);
            mixColumns(currBlock, mixMatrix);
            addRoundKey(currBlock, key, i);
        }
        subBytes(currBlock, sBox);
        shiftRows(currBlock);
        addRoundKey(currBlock, key, 14);


        for (int i = 0; i < 16; i++) {
            text[thread * 16 + i] = currBlock[i];
        }
    }
}


int main()
{
    ifstream ifs;
    ifs.open("plaintext.txt", std::ifstream::binary);
    if (!ifs) {
        cerr << "Cannot open the input file" << endl;
        exit(1);
    }
    ifs.seekg(0, ios::end);
    int infileLength = ifs.tellg();
    ifs.seekg(0, ios::beg);
    int blockNumber = infileLength / 16;
    int numberOfPadding = infileLength % 16;
    char* tempText = new char[(infileLength + numberOfPadding) * sizeof(char)];
    char  tempKey[32];
    uint8_t* text = new uint8_t[(infileLength + numberOfPadding) * sizeof(uint8_t)];
    uint8_t key[32];
    int textLen = infileLength + numberOfPadding;

    ifstream key_fp;
    key_fp.open("key.txt");
    if (!key_fp) {
        cerr << "Cannot open the key file" << endl;
        exit(1);
    }
    key_fp.seekg(0, ios::end);
    int keyfileLength = key_fp.tellg();
    key_fp.seekg(0, ios::beg);

    if (keyfileLength != 32) {
        printf("%s", "The key in key.txt needs to be 32 characters");
        return 0;
    }

    key_fp.read(tempKey, 32);

    ifs.read(tempText, infileLength);

    for (int i = 0; i < 32; i++) {
        key[i] = uint8_t(tempKey[i]);
    }

    for (int i = 0; i < infileLength; i++) {
        text[i] = uint8_t(tempText[i]);
    }

    for (int i = 0; i < numberOfPadding; i++) {
        text[infileLength - 1 + i] = uint8_t(numberOfPadding);
    }

    uint8_t expandedKey[240];
    keyExpansion(key, expandedKey);

    StartCounter();

    uint8_t* dev_text = 0;
    uint8_t* dev_key = 0;
    int textSize = (infileLength + numberOfPadding) * sizeof(uint8_t);
    int keySize = 240 * sizeof(uint8_t);
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&dev_text, textSize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_key, keySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_text, text, textSize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_key, expandedKey, keySize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    float totalThreads = (infileLength + numberOfPadding) / 16;
    int totalBlocks = ceil(totalThreads / 32);

    AESEncryption << <totalBlocks, 32 >> > (dev_text, dev_key, blockNumber);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(text, dev_text, textSize, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cout << "Execution Time: " << GetCounter() << " ms" << "\n";

    FILE* out_fp;
    out_fp = fopen("cipher.txt", "wb+");

    for (int j = 0; j < textLen; j++) {
        fprintf(out_fp, "%x ", text[j]);
    }

Error:
    hipFree(dev_text);
    hipFree(dev_key);

    return 0;
}
